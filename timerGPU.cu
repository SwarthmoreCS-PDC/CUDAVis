/* Copyright 2016-2018
 * Swarthmore College Computer Science, Swarthmore PA
 * T. Newhall, A. Danner
 */

#include "handle_cuda_error.h"
#include "timerGPU.h"
#include <cstdio>

using namespace std;

GPUTimer::GPUTimer() {
  HANDLE_ERROR(hipEventCreate(&startTime));
  HANDLE_ERROR(hipEventCreate(&stopTime));
}

GPUTimer::~GPUTimer() {
  HANDLE_ERROR(hipEventDestroy(startTime));
  HANDLE_ERROR(hipEventDestroy(stopTime));
}

void GPUTimer::start() { HANDLE_ERROR(hipEventRecord(startTime, 0)); }

void GPUTimer::stop() {
  HANDLE_ERROR(hipEventRecord(stopTime, 0));
  HANDLE_ERROR(hipEventSynchronize(stopTime));
}

void GPUTimer::print() { printf("GPU Time: %7.2f s\n", this->elapsed()); }

float GPUTimer::elapsed() {
  float elapsed_ms;
  this->stop();
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_ms, startTime, stopTime));
  return elapsed_ms;
}
